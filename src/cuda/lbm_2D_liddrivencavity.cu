#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <fstream>
#include <chrono>

// Definition of simulation parameters
constexpr int Q = 9;      // Number of directions (D2Q9)
constexpr int D = 2;      // Spatial dimension

// Inline functions to translate indices to linear indices
__device__ inline int idx_density(int i, int j, int NY) {
    return i * NY + j;
}
__device__ inline int idx_field(int i, int j, int k, int NY) {
    return (i * NY + j) * Q + k;
}
__device__ inline int idx_velocity(int i, int j, int d,int NY) {
    return (i * NY + j) * D + d;
}

__device__ inline double feq_func(int k, double rho, double ux, double uy, const double* w) {
    // Definition of D2Q9 directions
    int cx[Q] = { 0, 1, 0, -1,  0, 1, -1, -1, 1 };
    int cy[Q] = { 0, 0, 1,  0, -1, 1,  1, -1, -1 };

    double eu = cx[k] * ux + cy[k] * uy;
    double uv = ux * ux + uy * uy;
    return w[k] * rho * (1.0 + 3.0 * eu + 4.5 * eu * eu - 1.5 * uv);
}

// Kernel for collision and streaming step (compute)
__global__ void kernel_compute(double* f, double* f2, double* rho, double* rho2, double* u, double* u2, double tau_f, const double* w, int nx, int ny) {
    // Identify the cell managed by this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is inside the domain (skipping borders)
    if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
        int index = idx_density(i, j, ny);
        double local_rho = 0.0;
        double local_ux = 0.0;
        double local_uy = 0.0;

        int cx[Q] = { 0, 1, 0, -1, 0, 1, -1, -1, 1 };
        int cy[Q] = { 0, 0, 1, 0, -1, 1, 1, -1, -1 };

        // Loop over the Q directions to perform collision and streaming
        for (int k = 0; k < Q; k++) {
            // Determine the index of the source node (backward streaming)
            int ip = i - cx[k];
            int jp = j - cy[k];

            // Compute the indices for velocity, density, and distribution field in the source node
            int idx_comp = idx_density(ip, jp,ny);
            int idx_field_comp = idx_field(ip, jp, k,ny);

            // Load density and velocity values from the source node
            double rho_comp = rho[idx_comp];
            double u_x_comp = u[idx_velocity(ip, jp, 0,ny)];
            double u_y_comp = u[idx_velocity(ip, jp, 1,ny)];

            // Compute equilibrium value (feq)
            double feq = feq_func(k, rho_comp, u_x_comp, u_y_comp, w);

            // Apply collision
            double new_val = f[idx_field_comp] + (feq - f[idx_field_comp]) / tau_f;

            // Update distribution field for the current cell
            int idx_field_current = idx_field(i, j, k,ny);
            f2[idx_field_current] = new_val;

            // Accumulate contributions to density and velocity
            local_rho += new_val;
            local_ux += cx[k] * new_val;
            local_uy += cy[k] * new_val;
        }

        // Update density and velocity (normalized)
        rho2[index] = local_rho;
        u2[idx_velocity(i, j, 0,ny)] = local_ux / local_rho;
        u2[idx_velocity(i, j, 1,ny)] = local_uy / local_rho;
    }
}



// Kernel to apply boundary conditions for both left/right and top/bottom borders
__global__ void kernel_apply_boundary(double* f, double* rho, double* u, double u_lid, int nx, int ny, const double* w) {
    // Compute (i,j) indices based on 2D thread block configuration
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Linear index for density and velocity (each cell has D components)
    int idx = i * ny + j;

    // LEFT AND RIGHT BORDERS: for j in [1, NY-2]
    if (j >= 1 && j < ny - 1) {
        if (i == 0 || i == nx - 1) {
            // Set zero velocity at lateral borders
            u[idx * D + 0] = 0.0;
            u[idx * D + 1] = 0.0;

            // Set density equal to the inner neighbor's
            if (i == 0)
                rho[idx] = rho[1 * ny + j];       // copy from (1,j)
            else // i == NX-1
                rho[idx] = rho[(nx - 2) * ny + j];  // copy from (NX-2,j)

            // Update distribution field for each direction
            for (int k = 0; k < Q; k++) {
                // For left border (i==0), the inner node is (1,j)
                // For right border (i==NX-1), the inner node is (NX-2,j)
                int i_comp = (i == 0) ? 1 : (nx - 2);
                int idx_comp = i_comp * ny + j;
                // Get density and velocity of inner node
                double rho_comp = rho[idx_comp];
                double ux_comp = u[idx_comp * D + 0];
                double uy_comp = u[idx_comp * D + 1];

                // Compute feq for the current node (with zero velocity) and for the neighbor
                double feq_current = feq_func(k, rho[idx], 0.0, 0.0, w);
                double feq_comp = feq_func(k, rho_comp, ux_comp, uy_comp, w);

                // Updates are performed using the formula:
                // f(boundary, j, k) = feq(boundary, j, k) + f(inner node, j, k) - feq(inner node, j, k)
                int idx_field_boundary = (i * ny + j) * Q + k;
                int idx_field_comp = (i_comp * ny + j) * Q + k;
                f[idx_field_boundary] = feq_current + f[idx_field_comp] - feq_comp;
            }
        }
    }

    // TOP AND BOTTOM BORDERS: for any i, j==0 or j==NY-1
    if (i < nx) {
        if (j == 0 || j == ny - 1) {
            if (j == 0) {
                // Bottom border: zero velocity
                u[idx * D + 0] = 0.0;
                u[idx * D + 1] = 0.0;
                // Copy density from inner neighbor (node (i,1))
                rho[idx] = rho[i * ny + 1];
            }
            else {
                // Top border: set velocity to lid value
                u[idx * D + 0] = u_lid;
                u[idx * D + 1] = 0.0;
                // Copy density from inner neighbor (node (i,NY-2))
                rho[idx] = rho[i * ny + (ny - 2)];
            }

            // Update distribution field for each direction
            for (int k = 0; k < Q; k++) {
                int idx_field = (i * ny + j) * Q + k;
                if (j == 0) {
                    int idx_comp = (i * ny + 1);
                    double rho_comp = rho[idx_comp];
                    double ux_comp = u[idx_comp * D + 0];
                    double uy_comp = u[idx_comp * D + 1];
                    double feq_boundary = feq_func(k, rho[idx], 0.0, 0.0, w);
                    double feq_comp = feq_func(k, rho_comp, ux_comp, uy_comp, w);
                    int idx_field_comp = (i * ny + 1) * Q + k;
                    f[idx_field] = feq_boundary + f[idx_field_comp] - feq_comp;
                }
                else {  // j == NY-1
                    int idx_comp = (i * ny + (ny - 2));
                    double rho_comp = rho[idx_comp];
                    double ux_comp = u[idx_comp * D + 0];
                    double uy_comp = u[idx_comp * D + 1];
                    double feq_boundary = feq_func(k, rho[idx], u_lid, 0.0, w);
                    double feq_comp = feq_func(k, rho_comp, ux_comp, uy_comp, w);
                    int idx_field_comp = (i * ny + (ny - 2)) * Q + k;
                    f[idx_field] = feq_boundary + f[idx_field_comp] - feq_comp;
                }
            }
        }
    }
}

__global__ void kernel_init(double* rho, double* rho2,
    double* u, double* u2,
    double* f, double* F,
    int nx, int ny, double rho0, double u_lid, const double* w) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        int idx = idx_density(i, j,ny);

        // Initialize density and velocity fields
        rho[idx] = rho0;
        rho2[idx] = rho0;

        u[idx_velocity(i, j, 0,ny)] = 0.0;
        u[idx_velocity(i, j, 1,ny)] = 0.0;

        u2[idx_velocity(i, j, 0,ny)] = 0.0;
        u2[idx_velocity(i, j, 1,ny)] = 0.0;

        // Initialize distribution function with equilibrium values
        for (int k = 0; k < Q; k++) {
            double feq = feq_func(k, rho0, 0.0, 0.0, w);
            f[idx_field(i, j, k,ny)] = feq;
            F[idx_field(i, j, k,ny)] = feq;
        }
    }
}


constexpr int ITERATIONS_PER_PROGRESS_UPDATE = 100;
#include <chrono>
#include <iomanip> 

int main(int argc, char* argv[]) {

    if (argc != 7) {
        std::cerr << "Usage: " << argv[0]
                  << " <mesh_size> <time_steps> <reynolds> [output_dir]\n";
        return EXIT_FAILURE;
    }
    // Simulation parameters
    int NX = std::atoi(argv[1]);
    int NY = std::atoi(argv[2]);
    int MAX_STEPS = std::atoi(argv[3]);
    double Re = std::atof(argv[4]);
    int ITER_PER_FRAME = std::atoi(argv[5]);
    std::string out_dir = argv[6];
    const double u_lid = 0.5;
    int ITERATIONS_PER_FRAME = 200;
    
    const double dx = 1.0;
    double Lx = NY * dx;
    double nu = u_lid * Lx / Re;
    double tau_f = 3.0 * nu + 0.5;
    double rho0 = 1.0;

    // D2Q9 weights array
    double h_w[Q] = { 4.0 / 9, 1.0 / 9, 1.0 / 9, 1.0 / 9, 1.0 / 9,
                     1.0 / 36, 1.0 / 36, 1.0 / 36, 1.0 / 36 };

    // Compute field sizes
    size_t size_density = NX * NY * sizeof(double);
    size_t size_field = NX * NY * Q * sizeof(double);
    size_t size_velocity = NX * NY * D * sizeof(double);

    // Device memory allocation
    double* d_f, * d_f2, * d_rho, * d_rho2, * d_u, * d_u2, * d_w;
    hipMalloc(&d_f, size_field);
    hipMalloc(&d_f2, size_field);
    hipMalloc(&d_rho, size_density);
    hipMalloc(&d_rho2, size_density);
    hipMalloc(&d_u, size_velocity);
    hipMalloc(&d_u2, size_velocity);
    hipMalloc(&d_w, Q * sizeof(double));
    hipMemcpy(d_w, h_w, Q * sizeof(double), hipMemcpyHostToDevice);

    // Define block size and number
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((NX + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (NY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Initialization of fields
    kernel_init << <numBlocks, threadsPerBlock >> > (d_rho, d_rho2, d_u, d_u2, d_f, d_f2, NX, NY, rho0, u_lid, d_w);
    hipDeviceSynchronize();

    std::ofstream file_velocity(out_dir+"vel_data_cuda.txt");
    if (!file_velocity.is_open()) {
        std::cerr << "Error opening the file for velocity.\n";
        return 1;
    }
    file_velocity << NX << "\n" << NY << "\n";

    auto startTime = std::chrono::high_resolution_clock::now();

    for (int step = 1; step <= MAX_STEPS; step++) {
        // Collision and streaming step
        kernel_compute << <numBlocks, threadsPerBlock >> > (d_f, d_f2, d_rho, d_rho2, d_u, d_u2,
            tau_f, d_w, NX, NY);
        hipDeviceSynchronize();

        // Apply boundary conditions
        kernel_apply_boundary << <numBlocks, threadsPerBlock >> > (d_f, d_rho, d_u,
            u_lid, NX, NY, d_w);
        hipDeviceSynchronize();

        if (step % ITERATIONS_PER_FRAME == 0 || step == 1 || step == MAX_STEPS) {
            // Transfer results to host for saving to file 
            double* temp_vel = new double[NX * NY * D];
            hipMemcpy(temp_vel, d_u, size_velocity, hipMemcpyDeviceToHost);

            for (int j = 0; j < NY; ++j) {
                for (int i = 0; i < NX; ++i) {
                    double vx = temp_vel[(i * NY + j) * D + 0];
                    double vy = temp_vel[(i * NY + j) * D + 1];
                    double v = sqrt(vx * vx + vy * vy);
                    file_velocity << v << "\n";
                }
            }
            delete[] temp_vel;
        }

        // Swap pointers to prepare for next step
        double* tmp;
        tmp = d_f; d_f = d_f2; d_f2 = tmp;
        tmp = d_rho; d_rho = d_rho2; d_rho2 = tmp;
        tmp = d_u; d_u = d_u2; d_u2 = tmp;

        // Update the progress bar
        if (step % ITERATIONS_PER_PROGRESS_UPDATE == 0 || step == MAX_STEPS) {
            float progress = (static_cast<float>(step) / MAX_STEPS);
            auto currentTime = std::chrono::high_resolution_clock::now();
            auto elapsedTime = std::chrono::duration_cast<std::chrono::seconds>(currentTime - startTime).count();

            double estimatedTotalTime = elapsedTime / progress;
            int remainingTime = estimatedTotalTime - elapsedTime;

            progress *= 100;
            std::cout << "\rProgress: " << std::fixed << std::setprecision(2) << progress << "% completed "
                << "| Elapsed Time: " << elapsedTime << "s, "
                << "Remaining Time (estimated): " << static_cast<int>(remainingTime) << "s" << "        "
                << std::flush;

        }
    }

    file_velocity.close();

    // Free GPU and host memory
    hipFree(d_f);
    hipFree(d_f2);
    hipFree(d_rho);
    hipFree(d_rho2);
    hipFree(d_u);
    hipFree(d_u2);
    hipFree(d_w);

    return 0;
}
